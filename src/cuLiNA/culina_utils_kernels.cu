#include "hip/hip_runtime.h"
//
// Created by spades on 07/06/17.
//


#include <cuLiNA/culina_utils_kernels.cuh>
#include <cstdio>

__global__
void set_identity_matrix_kernel(double *d_matrix, int n_rows, int n_columns) {
    
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (i < n_rows && j < n_columns)
        if (i == j) {
            
            d_matrix[idx2c(i, j, n_rows)] = 1.0;
            //printf("%lf \n", d_matrix[IDX2C(i, j, n_rows)]);
            
        } else
            d_matrix[idx2c(i, j, n_rows)] = 0.0;
    
}

__global__
void culina_Dsumm_kernel(double *d_matrix1,
                         bool transpose_m1,
                         double alpha,
                         int n_rows,
                         int n_columns,
                         int ld,
                         double *d_matrix2,
                         bool transpose_m2,
                         double beta,
                         double *d_matrix_result,
                         double gamma) {
    
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (i < n_rows && j < n_columns) {
        
        double d_matrix1_element = 0;
        double d_matrix2_element = 0;
        double d_matrix_result_element = 0;
        
        if (alpha != 0)
            if (transpose_m1) d_matrix1_element = d_matrix1[idx2c(j, i, ld)];
            else d_matrix1_element = d_matrix1[idx2c(i, j, ld)];
        else d_matrix1_element = 0;
        
        if (beta != 0)
            if (transpose_m2) d_matrix2_element = d_matrix2[idx2c(j, i, ld)];
            else d_matrix2_element = d_matrix2[idx2c(i, j, ld)];
        else d_matrix2_element = 0;

        if(gamma != 0)
            d_matrix_result_element = gamma*d_matrix_result[idx2c(i,j,ld)] + d_matrix1_element*alpha + d_matrix2_element*beta;
        else d_matrix_result_element = d_matrix1_element*alpha + d_matrix2_element*beta;

//#if __CUDA_ARCH__ >= 200
//        printf("matrix3[%d, %d] = alpha*matrix1[%d, %d] + beta*matrix2[%d, %d] = %lf*%lf + %lf*%lf \n",
//            i,j,i,i,j,alpha,d_matrix1_element,beta,d_matrix2_element);
//#endif
        
        d_matrix_result[idx2c(i, j, ld)] = d_matrix_result_element;
        
    }
    
}

__global__
void culina_diagonal_Dmultiplication_kernel(double *d_matrix1,
                                            bool transpose_m1,
                                            double alpha,
                                            int n_rows_m1,
                                            int n_columns_m1,
                                            int ld_m1,
                                            double *d_matrix_diag,
                                            int ld_diag,
                                            double *d_matrix_result,
                                            double beta) {
    
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (i < n_rows_m1 && j < n_columns_m1){
    
        int idx_i = i;
        int idx_j = j;
        int ld_r = ld_m1;
        
        if(transpose_m1){ //xor swap procedure
            
            idx_i ^= idx_j;
            idx_j ^= idx_i;
            idx_i ^= idx_j;
            
            ld_r = n_columns_m1;
            
        }
        
        int m1_index = idx2c(i, j, ld_m1);
        int r_index =  idx2c(idx_i, idx_j, ld_r);
        int diag_index = idx2c(idx_j, 0, ld_diag);
    
        double tmp_var = 0;
        
        if(alpha != 0);
            tmp_var = alpha*d_matrix_diag[diag_index]*d_matrix1[m1_index];
        
//        printf("d_matrix_diag[%d, %d] = %lf \n", test_, test_, d_matrix_diag[diag_index]);
//        printf("d_matrix1[%d, %d] = %lf\n", r_i, r_j , d_matrix1[m1_index]);
        
        double tmp_var2 = 0;
        if(beta != 0)
            tmp_var2 = beta*d_matrix_result[m1_index];
        
        d_matrix_result[r_index] = tmp_var + tmp_var2;
//        printf("d_matrix_result[%d, %d] = %lf \n", r_i, r_j, d_matrix_result[r_index]);
    
    
//        printf("d_matrix_result[%d, %d] = d_matrix1[%d, %d]*d_matrix_diag[%d, %d] \n", idx_i, idx_j, i, j, idx_j, idx_j);
    }
    
    
}

__global__
extern void culina_Dskew_matrix3x3_operator_kernel(double *d_vector,
                                                   double alpha,
                                                   int n_rows_vector,
                                                   int ld_vector,
                                                   double *d_matrix_result,
                                                   int n_rows_result,
                                                   int n_columns_result,
                                                   int ld_result){
    
    int i = threadIdx.x;
    
    if (i < n_rows_result ){
        
            int k, l, idx;
        
            k = i;
            l = i+1;
            if(i == 2) l = 0;
        
            idx = i-1;
            if(i == 0) idx = 2;
        
            d_matrix_result[idx2c(i,i, ld_result)] = 0;
            d_matrix_result[idx2c(k,l, ld_result)] = -d_vector[idx]*alpha;
            d_matrix_result[idx2c(l,k, ld_result)] = d_vector[idx]*alpha;
        
    }
    
}

/***TODO create comments for this kernel
 *
 * m2.block(row_m2_init, col_m2_init, n_rows, n_cols) = alpha*op(m1.block(row_m1_init, col_m1_init, n_rows, n_cols))
 *
 * op(.) is the transpose operator used before copy
 *
 * */
__global__
void culina_Dblock_assingment_kernel(double *d_matrix1,
                                     bool transpose_m1,
                                     double alpha,
                                     int row_m1_init,
                                     int columns_m1_init,
                                     int ld_m1,
                                     double *d_matrix_result,
                                     int row_result_init,
                                     int columns_result_init,
                                     int ld_result,
                                     int n_rows,
                                     int n_columns) {
    
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    
    if(i < n_rows && j < n_columns){
    
        int d_matrix_idx = idx2c(row_m1_init+i, columns_m1_init+j, ld_m1);
        int d_result_idx = idx2c(row_result_init+i, columns_result_init+j, ld_result);
        
        if(transpose_m1) d_result_idx = idx2c(row_result_init+j, columns_result_init+i, ld_result);
        
        //printf("d_result_idx = %d - d_matrix_idx = %d \n", d_result_idx, d_matrix_idx );
        
        d_matrix_result[d_result_idx] = d_matrix1[d_matrix_idx]*alpha;
    
    }
    
}

/***TODO create comments for this kernel
 *
 * This kernel copies the weighted diagonal of a squared matrix to a column vector
 *
 * vector = diag(matrix)*alpha
 *
 * */
__global__
void culina_Ddiagonal_to_vector_kernel(double *d_matrix1,
                                       double alpha,
                                       int n_rows_m1,
                                       int n_columns_m1,
                                       int ld_m1,
                                       double *d_vector_result,
                                       int rows_result,
                                       int ld_result) {
    
    
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    
    //printf("%d" , i);
    
    if(i < rows_result && i < n_rows_m1 && j < n_columns_m1){
        
        d_vector_result[idx2c(i,j,ld_result)] = d_matrix1[idx2c(i,i,ld_m1)]*alpha;
    
    }
    
}